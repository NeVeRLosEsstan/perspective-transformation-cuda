#include "hip/hip_runtime.h"


#include <stdio.h>
#include <fstream>
#include <iostream>
#include <unistd.h>
#include <time.h>
#include <stdint.h>
using namespace std;

#define IMAGE_ROW 1300
#define IMAGE_COL 950

fstream file_coord;
fstream file_image;
fstream file_pt;

struct Image
{
	int R;
	int G;
	int B;
};

void normalize(float *matrix, float *I, float a, int i){
	for (int j = 0; j < 8; j++)
	{
		matrix[i*8+j] /= a;
		I[i*8+j] /= a;
	}
}

void print_matrix(float *matrix, int row, int col){
	for (int i = 0; i < row; i++)
	{	
		for (int j = 0; j < col; j++)
		{
			cout << matrix[i*row+j] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

void gauss(float *matrix, float *I, int i, int idx){
	for (int j = 0; j < 8; j++)
	{
		if (j!=idx)
		{
			float tmp = matrix[j*8+i];
			for (int k = 0; k <8; k++)
			{
				I[j*8+k] -= I[idx*8+k]*tmp;
				matrix[j*8+k] -= matrix[idx*8+k]*tmp;
			}
		}
	}
}

void homography(float *homo, float *matrix, float *target_coord, int *seq){
	float tmp;
	for (int i = 0; i < 8; i++)
	{
		tmp = 0;
		for (int j = 0; j < 8; j++)
		{
			tmp += matrix[seq[i]*8+j]*target_coord[j];
		}
		homo[i] = tmp;
	}	
	homo[8] = 1;
}

void perspective_trans(float *homo, Image *ori_image, Image *tar_image, float *x, float *y){
	float row_idx, col_idx;
	for (int i = 0; i < IMAGE_ROW; i++)
	{
		for (int j = 0; j < IMAGE_COL; j++)
		{
			row_idx = (homo[0]*j+homo[1]*i+homo[2])/(homo[6]*j+homo[7]*i+1);
			col_idx = (homo[3]*j+homo[4]*i+homo[5])/(homo[6]*j+homo[7]*i+1);
			if (row_idx<IMAGE_COL && col_idx<IMAGE_ROW && row_idx>=0 && col_idx>=0)
			{
				tar_image[int(col_idx)*IMAGE_COL+int(row_idx)].R = ori_image[i*IMAGE_COL+j].R;
				tar_image[int(col_idx)*IMAGE_COL+int(row_idx)].G = ori_image[i*IMAGE_COL+j].G;
				tar_image[int(col_idx)*IMAGE_COL+int(row_idx)].B = ori_image[i*IMAGE_COL+j].B;
			}
		}
	}
}

void output_img(Image *tar_image){
	file_pt.open("ptimage.txt", ios::out);
	for (int i = 0; i < IMAGE_ROW ; i++)
	{
		for (int j = 0; j < IMAGE_COL; j++)
		{
			file_pt << tar_image[i*IMAGE_COL+j].B << " "
					<< tar_image[i*IMAGE_COL+j].G << " "
					<< tar_image[i*IMAGE_COL+j].R << " "
					<< '\n';
		}
	}
}

void Init(float *orig_row_idx, float *orig_col_idx, float *targ_row_idx, 
	      float *targ_col_idx, float *I, int *flag, float *coord_matrix,
	      float *target_coord, Image *ori_image){

	file_coord.open("coord.txt", ios::in);
	file_image.open("image.txt", ios::in);

	for (int i = 0; i < 4; i++)
	{
		file_coord >> orig_row_idx[i];
		file_coord >> orig_col_idx[i];
	}

	for (int i = 0; i < 4; i++)
	{
		file_coord >> targ_row_idx[i];
		file_coord >> targ_col_idx[i];
	}

	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 8; j++)
		{
			if (i==j)
			{
				I[i*8+j] = 1;
			}
			else I[i*8+j] = 0;
			flag[i] = 0;
		}
	}

	for (int i = 0; i < 4; i++)
	{
		coord_matrix[2*i*8]    = orig_row_idx[i];
		coord_matrix[2*i*8+1]  = orig_col_idx[i];
		coord_matrix[2*i*8+2]  = 1;
		coord_matrix[2*i*8+3]  = 0;
		coord_matrix[2*i*8+4]  = 0;
		coord_matrix[2*i*8+5]  = 0;
		coord_matrix[2*i*8+6]  = -orig_row_idx[i]*targ_row_idx[i];
		coord_matrix[2*i*8+7]  = -orig_col_idx[i]*targ_row_idx[i];
		coord_matrix[2*i*8+8]  = 0;
		coord_matrix[2*i*8+9]  = 0;
		coord_matrix[2*i*8+10] = 0;
		coord_matrix[2*i*8+11] = orig_row_idx[i];
		coord_matrix[2*i*8+12] = orig_col_idx[i];
		coord_matrix[2*i*8+13] = 1;
		coord_matrix[2*i*8+14] = -orig_row_idx[i]*targ_col_idx[i];
		coord_matrix[2*i*8+15] = -orig_col_idx[i]*targ_col_idx[i];
		target_coord[2*i] = targ_row_idx[i];
		target_coord[2*i+1] = targ_col_idx[i];
	}

	for (int i = 0; i < IMAGE_ROW; i++)
	{
		for (int j = 0; j < IMAGE_COL; j++)
		{
			file_image >> ori_image[i*IMAGE_COL+j].B;
			file_image >> ori_image[i*IMAGE_COL+j].G;
			file_image >> ori_image[i*IMAGE_COL+j].R;
		}
	}
}

int main()
{
	double start_time, end_time;

	float* coord_matrix = new float[8*8];
	float* I = new float[8*8];
	float* homomatrix = new float[9];
	int* flag = new int[8];
	int* seq = new int[8];
	int idx;

	float* orig_row_idx = new float[4];
	float* orig_col_idx = new float[4];
	float* targ_row_idx = new float[4];
	float* targ_col_idx = new float[4];
	float* target_coord = new float[8];

	Image* ori_image = new Image[IMAGE_ROW*IMAGE_COL];
	Image* tar_image = new Image[IMAGE_ROW*IMAGE_COL];

	Init(orig_row_idx, orig_col_idx, targ_row_idx, targ_col_idx, I, 
		 flag, coord_matrix, target_coord, ori_image);

	start_time = clock();

	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 8; j++)
		{
			if (coord_matrix[j*8+i]!=0 && flag[j]==0)
			{
				idx = j;
				flag[j] = 1;
				seq[i] = j;
				break;
			}
		}
		float a = coord_matrix[idx*8+i];
		normalize(coord_matrix,I,a,idx);
		gauss(coord_matrix, I, i, idx);
	}

	homography(homomatrix, I, target_coord, seq);

	// print_matrix(I,8,8);
	// print_matrix(homomatrix,3,3);

	perspective_trans(homomatrix, ori_image, tar_image, targ_row_idx, targ_col_idx);
	
	end_time = clock();
 
	cout << "Execution Time: " << (end_time - start_time)/CLOCKS_PER_SEC << endl;

	output_img(tar_image);
	
	return 0;
}
