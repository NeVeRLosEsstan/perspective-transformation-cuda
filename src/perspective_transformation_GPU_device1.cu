#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <unistd.h>
#include <time.h>
#include <stdint.h>
#include "IndexSave.h"
#include <iomanip>
using namespace std;

#define IMAGE_ROW 1300
#define IMAGE_COL 950
#define BLOCK_SIZE 16

fstream file_coord;
fstream file_image;
fstream file_pt;

struct Image
{
	int R;
	int G;
	int B;
};

__global__ void normalize(float *matrix, float *I, int i){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	float a = matrix[i*8+i];
	if (blockIdx.x==i)
	{
		matrix[idx] /= a;
		I[idx] /= a;
	}
}

__global__ void gauss(float *matrix, float *I, int i){
	if (blockIdx.x!=i)
	{
		I[blockIdx.x*8+threadIdx.x] -= I[i*8+threadIdx.x]*matrix[blockIdx.x*8+i];
		if (threadIdx.x!=i)
		{
			matrix[blockIdx.x*8+threadIdx.x] -= matrix[i*8+threadIdx.x]*matrix[blockIdx.x*8+i];
		}
	}
}

__global__ void swap(float *matrix, float *I, int i){
	float tmp;
	for (int j = i; j < 8; j++)
	{
		if (matrix[i*8+i]==0 && matrix[j*8+i]!=0)
		{
			tmp = matrix[i*8+threadIdx.x];
			matrix[i*8+threadIdx.x] = matrix[j*8+threadIdx.x];
			matrix[j*8+threadIdx.x] = tmp;

			tmp = I[i*8+threadIdx.x];
			I[i*8+threadIdx.x] = I[j*8+threadIdx.x];
			I[j*8+threadIdx.x] = tmp;
		}
	}
}

__global__ void homography(float *homo, float *matrix, float *target_coord){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
	float tmp = 0;
	if (idx<8)
	{
		for (int i = 0; i < 8; i++)
		{
			tmp += matrix[idx*8+i]*target_coord[i];
		}
		homo[idx] = tmp;
	}

	homo[8] = 1;
}

__global__ void perspective_trans(float *homo, Image *ori_image, Image *tar_image){
	int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col<IMAGE_COL && row<IMAGE_ROW)
	{
		float row_idx, col_idx;
	
		row_idx = (homo[0]*col+homo[1]*row+homo[2])/(homo[6]*col+homo[7]*row+1);
		col_idx = (homo[3]*col+homo[4]*row+homo[5])/(homo[6]*col+homo[7]*row+1);
	
		if (row_idx<IMAGE_COL && col_idx<IMAGE_ROW && row_idx>=0 && col_idx>=0)
		{
			tar_image[int(col_idx)*IMAGE_COL+int(row_idx)].R = ori_image[row*IMAGE_COL+col].R;
			tar_image[int(col_idx)*IMAGE_COL+int(row_idx)].G = ori_image[row*IMAGE_COL+col].G;
			tar_image[int(col_idx)*IMAGE_COL+int(row_idx)].B = ori_image[row*IMAGE_COL+col].B;
		}
	}
}

void print_matrix(float *matrix, int row, int col){
	for (int i = 0; i < row; i++)
	{	
		for (int j = 0; j < col; j++)
		{
			cout << matrix[i*row+j] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

void output_img(Image *tar_image){
	file_pt.open("ptimage_gpu.txt", ios::out);
	for (int i = 0; i < IMAGE_ROW ; i++)
	{
		for (int j = 0; j < IMAGE_COL; j++)
		{
			file_pt << tar_image[i*IMAGE_COL+j].B << " "
					<< tar_image[i*IMAGE_COL+j].G << " "
					<< tar_image[i*IMAGE_COL+j].R << " "
					<< '\n';
		}
	}
}

void Init(float *orig_row_idx, float *orig_col_idx, float *targ_row_idx, 
	      float *targ_col_idx, float *I, float *coord_matrix,
	      float *target_coord, Image *ori_image){

	file_coord.open("coord.txt", ios::in);
	file_image.open("image.txt", ios::in);

	for (int i = 0; i < 4; i++)
	{
		file_coord >> orig_row_idx[i];
		file_coord >> orig_col_idx[i];
	}

	for (int i = 0; i < 4; i++)
	{
		file_coord >> targ_row_idx[i];
		file_coord >> targ_col_idx[i];
	}

	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 8; j++)
		{
			if (i==j)
			{
				I[i*8+j] = 1;
			}
			else I[i*8+j] = 0;
		}
	}

	for (int i = 0; i < 4; i++)
	{
		coord_matrix[2*i*8]    = orig_row_idx[i];
		coord_matrix[2*i*8+1]  = orig_col_idx[i];
		coord_matrix[2*i*8+2]  = 1;
		coord_matrix[2*i*8+3]  = 0;
		coord_matrix[2*i*8+4]  = 0;
		coord_matrix[2*i*8+5]  = 0;
		coord_matrix[2*i*8+6]  = -orig_row_idx[i]*targ_row_idx[i];
		coord_matrix[2*i*8+7]  = -orig_col_idx[i]*targ_row_idx[i];
		coord_matrix[2*i*8+8]  = 0;
		coord_matrix[2*i*8+9]  = 0;
		coord_matrix[2*i*8+10] = 0;
		coord_matrix[2*i*8+11] = orig_row_idx[i];
		coord_matrix[2*i*8+12] = orig_col_idx[i];
		coord_matrix[2*i*8+13] = 1;
		coord_matrix[2*i*8+14] = -orig_row_idx[i]*targ_col_idx[i];
		coord_matrix[2*i*8+15] = -orig_col_idx[i]*targ_col_idx[i];
		target_coord[2*i] = targ_row_idx[i];
		target_coord[2*i+1] = targ_col_idx[i];
	}

	for (int i = 0; i < IMAGE_ROW; i++)
	{
		for (int j = 0; j < IMAGE_COL; j++)
		{
			file_image >> ori_image[i*IMAGE_COL+j].B;
			file_image >> ori_image[i*IMAGE_COL+j].G;
			file_image >> ori_image[i*IMAGE_COL+j].R;
		}
	}
}

int main()
{
	float elapsedTime;
	hipEvent_t start,stop;
	hipEventCreate (&start);
	hipEventCreate (&stop);

	float* coord_matrix = new float[8*8];
	float* I = new float[8*8];
	float* homomatrix = new float[9];
	float* d_coord_matrix;
	float* d_I;
	float* d_homomatrix;

	float* orig_row_idx = new float[4];
	float* orig_col_idx = new float[4];
	float* targ_row_idx = new float[4];
	float* targ_col_idx = new float[4];
	float* target_coord = new float[8];
	float* d_target_coord;

	Image* ori_image = new Image[IMAGE_ROW*IMAGE_COL];
	Image* tar_image = new Image[IMAGE_ROW*IMAGE_COL];
	Image* d_ori_image;
	Image* d_tar_image;

	Init(orig_row_idx, orig_col_idx, targ_row_idx, targ_col_idx, I, 
		 coord_matrix, target_coord, ori_image);

	hipEventRecord(start, 0);

	hipMalloc((void**)& d_coord_matrix, 8*8 * sizeof(float));
	hipMalloc((void**)& d_I, 8*8 * sizeof(float));

	hipMalloc((void**)& d_homomatrix, 9 * sizeof(float));
	hipMalloc((void**)& d_target_coord, 8 * sizeof(float));
	hipMalloc((void**)& d_ori_image, IMAGE_ROW*IMAGE_COL * sizeof(Image));
	hipMalloc((void**)& d_tar_image, IMAGE_ROW*IMAGE_COL * sizeof(Image));

	hipMemcpy(d_coord_matrix, coord_matrix, 8*8 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_I, I, 8*8 * sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(d_target_coord, target_coord, 8 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_ori_image, ori_image, IMAGE_ROW*IMAGE_COL * sizeof(Image), hipMemcpyHostToDevice);
	
	
	for (int i = 0; i < 8; i++)
	{
		dim3 dimBlock(8);
		dim3 dimGrid(8);
		swap << < dimGrid, dimBlock >> > (d_coord_matrix, d_I, i);
		normalize << < dimGrid, dimBlock >> > (d_coord_matrix, d_I, i);
		gauss << < dimGrid, dimBlock >> > (d_coord_matrix, d_I, i);
	}

	dim3 homodimBlock(8);
	dim3 homodimGrid(1);
	homography << < homodimGrid, homodimBlock >> > (d_homomatrix, d_I, d_target_coord/*, dInd*/);
	
	dim3 ptdimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 ptdimGrid((IMAGE_COL + BLOCK_SIZE - 1)/BLOCK_SIZE, (IMAGE_ROW + BLOCK_SIZE - 1)/BLOCK_SIZE);
	perspective_trans << < ptdimGrid, ptdimBlock >> > (d_homomatrix, d_ori_image, d_tar_image);

	hipDeviceSynchronize();
	hipMemcpy(tar_image, d_tar_image, IMAGE_ROW*IMAGE_COL * sizeof(Image), hipMemcpyDeviceToHost);
	
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "Execution Time: " << (elapsedTime)/CLOCKS_PER_SEC << endl;

	output_img(tar_image);

	return 0;
}
